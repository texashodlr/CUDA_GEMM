#include "hip/hip_runtime.h"
#include "cudaLib.cuh"
#include <algorithm>
#define GEMM_TILE_SIZE 32

void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	//	Insert code here
	std::cout << "Lazy, you are!\n";
	std::cout << "Write code, you must\n";

	return 0;
}

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	//	Insert code here
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 3.14159f;

	std::cout << "Sneaky, you are ...\n";
	std::cout << "Compute pi, you must!\n";
	return approxPi;
}

int runGpuMedianFilter (std::string imgPath, std::string outPath, MedianFilterArgs args) {
	
	std::cout << "Lazy, you are! ... ";
	std::cout << "Filter pixels, you must! ... ";

	return 0;
}

int medianFilter_gpu (uint8_t inPixels, ImageDim imgDim, 
	uint8_t outPixels, MedianFilterArgs args) {

	return 0;
}


int runGpuConv (int argc, char ** argv) {

	TensorShape iShape = AlexL1_InShape;
	TensorShape fShape = AlexL1_FilterShape;
	ConvLayerArgs convArgs = AlexL1_ConvArgs;

	TensorShape oShape;
			
	uint64_t errorCount = evaluateGpuConv(iShape, fShape, oShape, convArgs);
	std::cout << "Found " << errorCount << " / " << tensorSize(oShape) << " errors \n";
	return 0;
}

uint64_t executeGpuConv(TensorShape iShape, TensorShape fShape,
	TensorShape& oShape, ConvLayerArgs args) {

	oShape.height = (iShape.height + 2 * args.padH - fShape.height) / args.strideH + 1;
	oShape.width = (iShape.width + 2 * args.padW - fShape.width) / args.strideW + 1;
	oShape.channels = (fShape.count);
	oShape.count = 1;

	float* h_in = nullptr;
	float* h_filter = nullptr;
	float* h_bias = nullptr;
	float* h_out = nullptr;

	int retVal;
	retVal = makeTensor(&h_in, iShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		return 99;
	}
	retVal = makeTensor(&h_filter, fShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		return 99;
	}
	retVal = makeVector(&h_bias, oShape.channels);
	if (retVal != 0) {
		std::cout << "Unable to make vector \n";
		return 99;
	}

	//std::cout << "OutShape : " << oShape << " \n";
	h_out = (float*)malloc(tensorSize(oShape) * sizeof(float));

	/*CUDA Malloc for in, out, filter and bias*/

	float* d_in, * d_filter, * d_bias, * d_out;
	hipMalloc(&d_in, tensorSize(iShape) * sizeof(float));
	hipMalloc(&d_filter, tensorSize(fShape) * sizeof(float));
	hipMalloc(&d_bias, (oShape.channels) * sizeof(float));
	hipMalloc(&d_out, tensorSize(oShape) * sizeof(float));

	/*CUDA Memcpy for in, filter and bias*/
	hipMemcpy(d_in, h_in, tensorSize(iShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_filter, h_filter, tensorSize(fShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_bias, h_bias, (oShape.channels) * sizeof(float), hipMemcpyHostToDevice);

	/*Tile Sizing Dynamically based on my 4070 (49KB/SM)*/
	const int max_floats = 49152 / (4 * iShape.channels); //Sized for my 4070/v100
	const int max_tile_dim = std::floor(std::sqrt(max_floats)) - fShape.height + 1;
	const int oShape_dims = std::min(oShape.height, oShape.width);
	//std::cout << "Sizing options: Max_tile_dim: " << max_tile_dim << " | oShape_dims: " << oShape_dims << "\n";
	int TILE_SIZE = std::min(max_tile_dim, oShape_dims);
	
	if (TILE_SIZE > 15) {
		TILE_SIZE = 6;
	}

	/*Block and Grid Dims*/
	dim3 blockDim(TILE_SIZE, TILE_SIZE, 1);
	dim3 gridDim((oShape.width + TILE_SIZE - 1) / TILE_SIZE, (oShape.height + TILE_SIZE - 1) / TILE_SIZE, oShape.channels);

	int shared_window_height = ((TILE_SIZE - 1) * args.strideH) + fShape.height; // 31
	int shared_window_width = ((TILE_SIZE - 1) * args.strideW) + fShape.width;  // 31

	size_t sharedMemSize = ((iShape.channels * (shared_window_height) * (shared_window_width))) * sizeof(float); // 4B*(3*31*31) = 11,5KB


	/*ConvLayer Kernel Call*/
	//convLayer_gpu << <gridDim, blockDim >> > (d_in, iShape, d_filter, fShape, d_bias, d_out, oShape, args);
	//std::cout << "\n\n GPU Starting!\n\n\n";
	//std::cout << "Bias[0] = " << h_bias[0] << "\n";
	//std::cout << "Memory Size: " << sharedMemSize << " Bytes! \n";
	//convLayer_gpu << <gridDim, blockDim >> > (d_in, iShape, d_filter, fShape, d_bias, d_out, oShape, args);

	auto start = std::chrono::high_resolution_clock::now();
	//convLayer_gpu << <gridDim, blockDim >> > (d_in, iShape, d_filter, fShape, d_bias, d_out, oShape, args);
	convLayer_gpu_SM_DM_v3 << <gridDim, blockDim, sharedMemSize >> > (d_in, iShape, d_filter, fShape, d_bias, d_out, oShape, args, TILE_SIZE);
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "GPU execution time: " << elapsed.count() << " seconds\n";

	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Kernel Launch Error: " << hipGetErrorString(err) << "\n";
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		std::cout << "CUDA Error: " << hipGetErrorString(err) << "\n";
	}

	/*CUDA memcpy for d_out to d_in*/
	hipMemcpy(h_out, d_out, tensorSize(oShape) * sizeof(float), hipMemcpyDeviceToHost);

	//std::cout << "\n Comparing CPU and GPU now...\n";

	float* cpu_out = executeCpuConv2(iShape, fShape, oShape, args);
	uint64_t verify_errors = verifyVector_convLayer(cpu_out, h_out, (oShape.height * oShape.width * oShape.channels));
	std::cout << "\nFound " << verify_errors << " Errors...\n";

	/* hipFree() functions */
	hipFree(d_in);
	hipFree(d_filter);
	hipFree(d_bias);
	hipFree(d_out);

	/*CPU Free*/
	free(h_in);
	free(h_filter);
	free(h_bias);
	free(h_out);
	return verify_errors;
}

uint64_t evaluateGpuConv (TensorShape iShape, TensorShape fShape, 
	TensorShape & oShape, ConvLayerArgs args) {

	uint64_t errorCount = 0;

	errorCount = executeGpuConv(iShape, fShape, oShape, args);

	#ifndef CONV_CHECK_DISABLE
		//	STUDENT: Verify number of errors in ouput matrix generated by convLayer_gpu
		//	STUDENT: Compare results with CPU output
		//	STUDENT: Return error count
	errorCount = executeGpuConv(iShape, fShape, oShape, args);

	#endif

	return errorCount;
}

__global__ void convLayer_gpu(float* input, TensorShape iShape, float* filter,
	TensorShape fShape, float* bias, float* output, TensorShape oShape, ConvLayerArgs args) {

	/*Coordinates*/
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int channels = blockIdx.z;

	if (col < oShape.width && row < oShape.height) {
		uint32_t out_idx = ((channels * oShape.height + row) * oShape.width + col);
		output[out_idx] = bias[channels];
		//CPU Code For-loop repeat for the filter window//
		for (uint32_t i = 0; i < fShape.height; ++i) {
			for (uint32_t j = 0; j < fShape.width; ++j) {
				for (uint32_t k = 0; k < fShape.channels; ++k) {
					uint32_t in_h = args.strideH * row + i;
					uint32_t in_w = args.strideW * col + j;

					if (in_h < iShape.height && in_w < iShape.width) {
						uint32_t in_idx = (k * iShape.height + in_h) * iShape.width + in_w;
						uint32_t filter_idx = ((channels * fShape.channels + k) * fShape.height + i) * fShape.width + j;

						output[out_idx] += input[in_idx] * filter[filter_idx];
					}
				}
			}
		}
		if (args.activation) {
			output[out_idx] = fmaxf(0.0f, output[out_idx]);
		}
	}
}

__global__ void convLayer_gpu_SM_DM_v3(float* input, TensorShape iShape, float* filter,
	TensorShape fShape, float* bias, float* output, TensorShape oShape, ConvLayerArgs args, int TILE_SIZE) {

	extern __shared__ float tile[];
	
	const int tile_height = ((TILE_SIZE - 1) * args.strideH) + fShape.height;  // 6-1*4+11 = 31 
	const int tile_width  = ((TILE_SIZE - 1) * args.strideW) + fShape.width;   // " "		 = 31
	const int tile_depth = iShape.channels;									   //			 = 3
	
	/*Shared Memory Sizing: 31x31x3 = 2,883 elements, 961 elements per channel*/

	int tidx = threadIdx.x;
	int tidy = threadIdx.y;

	int out_x = blockIdx.x * blockDim.x + tidx;
	int out_y = blockIdx.y * blockDim.y + tidy;
	int out_z = blockIdx.z;

	int threadblock_thread_id = tidy * blockDim.x + tidx; // 0:35

	int base_x = blockIdx.x * blockDim.x * args.strideW; // 0:9*6*4 = 0 --> 216
	int base_y = blockIdx.y * blockDim.y * args.strideH; // 0:9*6*4 = 0 --> 216

	/*Load full 31x31 * 3 tile cooperatively*/
	/*First 12 threads load 80 elements per*/

	//int coop_tid = tidy * TILE_SIZE + tidx;
	int total_threads = TILE_SIZE * TILE_SIZE;			  // 6*6  = 36
	//int threads_per_channel = total_threads / tile_depth; // 36/3 = 12
	int elements_per_channel = (tile_height * tile_width); 
	int total_elements = tile_depth * elements_per_channel;

	//if (threadblock_thread_id == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0) {
		//printf("Block (0, 0, 0): Expected shared memory = %d bytes\n", total_elements * 4);}

	/*Cooperative Loading*/
	for (int idx = threadblock_thread_id; idx < total_elements; idx += total_threads) {
		int c = idx / elements_per_channel; //0:2
		int coop_idx = idx % elements_per_channel;
		int coop_y = coop_idx / tile_width; // row in shared tile (0 to tile_height-1)
		int coop_x = coop_idx % tile_width; // col in shared tile (0 to tile_width-1)

		// Compute the corresponding global input coordinates.
		// Incorporate any padding in the index calculation if needed.
		int input_x = base_x + coop_x; // PAD ?
		int input_y = base_y + coop_y; // PAD ?
		//int input_x = base_x + coop_x + args.padW;
		//int input_y = base_y + coop_y + args.padH;

		int shared_idx = (c * tile_height * tile_width) + coop_y * tile_width + coop_x;
		//printf("Channel: %d | Input_x: %d | Input_y: %d\n", c, input_x, input_y);
		if (input_x >= 0 && input_x < iShape.width &&
			input_y >= 0 && input_y < iShape.height) {
			int global_idx = (c * iShape.height + input_y) * iShape.width + input_x;
			tile[shared_idx] = input[global_idx];
			//tile[shared_idx] = 1.0f;
		}
		else {
			tile[shared_idx] = 0.0f;  // Handle boundaries (zero padding)
			
		}

	}

	__syncthreads();

	// Debug: Confirm execution
	//if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadblock_thread_id == 0) {
		//printf("Block (0, 0, 0): Loading completed\n");	}

	// Output computing, somewhat similar to CPU/GM
	if (out_x < oShape.width && out_y < oShape.height) {
		uint32_t out_idx = (out_z * oShape.height + out_y) * oShape.width + out_x;
		float shared_sum = bias[out_z];

		for (uint32_t k = 0; k < fShape.channels; ++k) {
			float channel_sum = 0.0f;
			for (uint32_t i = 0; i < fShape.height; ++i) {
				for (uint32_t j = 0; j < fShape.width; ++j) {
					int in_x = (out_x)*args.strideW + j;
					int in_y = out_y * args.strideH + i;
					int tile_x = in_x - base_x;
					int tile_y = in_y - base_y;
					
					//Bounds check with input shape
					if (in_x < iShape.width && in_y < iShape.height) {
						uint32_t shared_idx = k * tile_height * tile_width + tile_y * tile_width + tile_x;
						uint32_t filter_idx = (out_z * fShape.channels + k) * fShape.height * fShape.width + i * fShape.width + j;
						channel_sum += tile[shared_idx] * filter[filter_idx];
					}
				}
			}
			shared_sum += channel_sum;
		}
		output[out_idx] = shared_sum;
		//Relu again-lu
		if (args.activation) {
			output[out_idx] = fmaxf(0.0f, output[out_idx]);
		}
	}

}

int runGpuGemm (int argc, char ** argv) {

	//executeCpuGemm_v1(aShape, bShape, cShape, args, BatchSize);
	uint32_t BatchSize = 3;
	TensorShape aShape = { BatchSize, 1, 1, 4096 };
	TensorShape bShape = { 1, 1, 4096, 4096 };
	TensorShape cShape;
	GemmLayerArgs args = { 2, 2, 1 };

	std::cout << "Executing GPU COPY GEMM with BatchSize: " << BatchSize << "\n";
	float* gpu_copy = evaluateGpuGemm_copy_speed2(aShape, bShape, cShape, args, BatchSize);
	//evaluateGpuGemm_copy_speed(aShape, bShape, cShape, args, BatchSize);

	std::cout << "Executing GPU UVM GEMM with BatchSize: " << BatchSize << "\n";
	float* gpu_uvm = evaluateGpuGemm_uvm2(aShape, bShape, cShape, args, BatchSize);
	
	float* cpu_gemm = executeCpuGemm_v3(aShape, bShape, cShape, args);
	int errorCount = 0;

	errorCount = verifyVector_gemm(gpu_copy, gpu_uvm, cpu_gemm, (aShape.count*aShape.channels*aShape.height*bShape.width));
	std::cout << "\nFound " << errorCount << " Errors...\n";
	return errorCount;
}


__global__ void gemmLayer_gpu_speed(float* a, TensorShape aShape,	float* b, TensorShape bShape,
	float* c, TensorShape cShape) {
	extern __shared__ float shared[];
	float* Mds = shared;
	float* Nds = shared + GEMM_TILE_SIZE * GEMM_TILE_SIZE;
	int bx = blockIdx.x;
	//int by = blockIdx.y;
	int bz = blockIdx.z;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Row		= bz;
	int Col		= bx * GEMM_TILE_SIZE + tx;
	float pVal = 0;
	int Width = aShape.width; //Inner Mat Dim
	if (Row < aShape.count) {
		for (int p_idx = 0; p_idx < (Width + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE; ++p_idx) {

			if (ty < GEMM_TILE_SIZE && tx < GEMM_TILE_SIZE && Col < bShape.width) {
				int b_row = p_idx * GEMM_TILE_SIZE + ty;
				if (b_row < bShape.height) {
					Nds[ty * GEMM_TILE_SIZE + tx] = b[b_row * bShape.width + Col];
				}
				else {
					Nds[ty * GEMM_TILE_SIZE + tx] = 0.0f;
				}
			}
			//Loading A into Mds
			if (ty < GEMM_TILE_SIZE && (p_idx * GEMM_TILE_SIZE + tx) < aShape.width) {
				Mds[ty*GEMM_TILE_SIZE+tx] = a[Row * Width + p_idx * GEMM_TILE_SIZE + tx];
			}
			else {
				Mds[ty * GEMM_TILE_SIZE + tx] = 0.0f;
			}
			__syncthreads();
			for (int k = 0; k < GEMM_TILE_SIZE; ++k) {
				pVal += Mds[ty * GEMM_TILE_SIZE + k] * Nds[k*GEMM_TILE_SIZE+tx];
			}
			__syncthreads();
		}
		if (ty == 0 && Col < cShape.width) {
			c[Row * cShape.width + Col] = pVal;
		}
	}
	
}

__global__ void gemmLayer_gpu_v2(float* a, TensorShape aShape, float* b, TensorShape bShape,
	float* c, TensorShape cShape){
	__shared__ float Mds[GEMM_TILE_SIZE][GEMM_TILE_SIZE];
	__shared__ float Nds[GEMM_TILE_SIZE][GEMM_TILE_SIZE];

	int bx = blockIdx.x;
	//int by = blockIdx.y;
	int bz = blockIdx.z; //Batching idx 0:2
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Row = bz;
	int Col = bx * GEMM_TILE_SIZE + tx;
	float pVal = 0;
	int Width = aShape.width; //Inner Mat Dim
	if (Row < aShape.count) {
		for (int p_idx = 0; p_idx < (Width + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE; ++p_idx) {
			//Loading A into Mds
			if (ty < GEMM_TILE_SIZE && (p_idx * GEMM_TILE_SIZE + tx) < aShape.width) {
				Mds[ty][tx] = a[Row * Width + p_idx * GEMM_TILE_SIZE + tx];
			}
			else {
				Mds[ty][tx] = 0.0f;
			}
			//Loading B into Nds
			if (Col < bShape.width && (p_idx * GEMM_TILE_SIZE + ty) < bShape.height) {
				Nds[ty][tx] = b[(p_idx * GEMM_TILE_SIZE + ty) * bShape.width + Col];
			}
			else {
				Nds[ty][tx] = 0.0f;
			}
			__syncthreads();
			for (int k = 0; k < GEMM_TILE_SIZE; ++k) {
				pVal += Mds[ty][k] * Nds[k][tx];
			}
			__syncthreads();

		}
		if (ty == 0 && Col < cShape.width) {
			c[Row * cShape.width + Col] = pVal;
		}
	}
}

__global__ void gemmLayer_gpu(float* a, TensorShape aShape, float* b, TensorShape bShape,
	float* c, TensorShape cShape) {
	__shared__ float Mds[GEMM_TILE_SIZE][GEMM_TILE_SIZE];
	__shared__ float Nds[GEMM_TILE_SIZE][GEMM_TILE_SIZE];
	int bx = blockIdx.x;
	//int by = blockIdx.y;
	int bz = blockIdx.z; //Batching idx 0:2
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Row = bz;
	int Col = bx * GEMM_TILE_SIZE + tx;
	float pVal = 0;
	int Width = aShape.width; //Inner Mat Dim
	if (Row < aShape.count) {
		for (int p_idx = 0; p_idx < (Width + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE; ++p_idx) {
			//Loading A into Mds
			if (ty < GEMM_TILE_SIZE && (p_idx * GEMM_TILE_SIZE + tx) < aShape.width) {
				Mds[ty][tx] = a[Row * Width + p_idx * GEMM_TILE_SIZE + tx];
			}
			else {
				Mds[ty][tx] = 0.0f;
			}
			//Loading B into Nds
			if (Col < bShape.width && (p_idx * GEMM_TILE_SIZE + ty) < bShape.height) {
				Nds[ty][tx] = b[(p_idx * GEMM_TILE_SIZE + ty) * bShape.width + Col];
			}
			else {
				Nds[ty][tx] = 0.0f;
			}
			__syncthreads();
			for (int k = 0; k < GEMM_TILE_SIZE; ++k) {
				pVal += Mds[ty][k] * Nds[k][tx];
			}
			__syncthreads();
		}
		if (ty == 0 && Col < cShape.width) {
			c[Row * cShape.width + Col] = pVal;
		}
	}
}

int evaluateGpuGemm_copy_speed(TensorShape aShape, TensorShape bShape,
	TensorShape& cShape, GemmLayerArgs args, uint32_t BatchSize) {

	cShape.height = aShape.height;
	cShape.width = bShape.width;
	cShape.channels = aShape.channels;
	cShape.count = BatchSize;

	float* h_a = nullptr;
	float* h_b = nullptr;

	int retVal;
	retVal = makeTensor(&h_a, aShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		return -1;
	}
	retVal = makeTensor(&h_b, bShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		return -1;
	}

	/*C (out) Array initialization*/
	float* h_c = (float*)malloc(tensorSize(cShape) * sizeof(float));

	/*CUDA Malloc for in, out, filter and bias*/

	float* d_a, * d_b, * d_c;
	hipMalloc(&d_a, tensorSize(aShape) * sizeof(float));
	hipMalloc(&d_b, tensorSize(bShape) * sizeof(float));
	hipMalloc(&d_c, tensorSize(cShape) * sizeof(float));

	/*CUDA Memcpy for in, filter and bias*/
	hipMemcpy(d_a, h_a, tensorSize(aShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, tensorSize(bShape) * sizeof(float), hipMemcpyHostToDevice);


	/*Block and Grid Dims*/
	dim3 blockDim(GEMM_TILE_SIZE, GEMM_TILE_SIZE);
	dim3 gridDim((cShape.width + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE, 1, cShape.count); // 128x1x3
	size_t sharedMemSize = 2 * GEMM_TILE_SIZE * GEMM_TILE_SIZE * sizeof(float);

	std::cout << "'Speed-up' COPY GPU Starting!\n";
	hipEvent_t start1, stop1;
	hipEventCreate(&start1); hipEventCreate(&stop1);
	hipEventRecord(start1);
	std::cout << "\tMemory Size: " << sharedMemSize << " Bytes! \n";
	auto start = std::chrono::high_resolution_clock::now();
	gemmLayer_gpu_speed << <gridDim, blockDim, sharedMemSize >> > (d_a, aShape, d_b, bShape, d_c, cShape);
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "GPU execution time: " << elapsed.count() << " seconds\n";
	//gemmLayer_gpu_v2 << <gridDim, blockDim >> > (d_a, aShape, d_b, bShape, d_c, cShape);
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	float ms; hipEventElapsedTime(&ms, start1, stop1);
	std::cout << "'Speed-up' COPY Batch " << BatchSize << " Time: " << ms << " ms\n";
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Kernel Launch Error: " << hipGetErrorString(err) << "\n";
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		std::cout << "CUDA Error: " << hipGetErrorString(err) << "\n";
	}

	hipMemcpy(h_c, d_c, tensorSize(cShape) * sizeof(float), hipMemcpyDeviceToHost);

	/* hipFree() functions */
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	/*CPU Free*/
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}

float* evaluateGpuGemm_copy_speed2(TensorShape aShape, TensorShape bShape,
	TensorShape& cShape, GemmLayerArgs args, uint32_t BatchSize) {

	cShape.height = aShape.height;
	cShape.width = bShape.width;
	cShape.channels = aShape.channels;
	cShape.count = BatchSize;

	float* h_a = nullptr;
	float* h_b = nullptr;

	int retVal;
	retVal = makeTensor(&h_a, aShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		//return -1;
	}
	retVal = makeTensor(&h_b, bShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		//return -1;
	}

	/*C (out) Array initialization*/
	float* h_c = (float*)malloc(tensorSize(cShape) * sizeof(float));

	/*CUDA Malloc for in, out, filter and bias*/

	float* d_a, * d_b, * d_c;
	hipMalloc(&d_a, tensorSize(aShape) * sizeof(float));
	hipMalloc(&d_b, tensorSize(bShape) * sizeof(float));
	hipMalloc(&d_c, tensorSize(cShape) * sizeof(float));

	/*CUDA Memcpy for in, filter and bias*/
	hipMemcpy(d_a, h_a, tensorSize(aShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, tensorSize(bShape) * sizeof(float), hipMemcpyHostToDevice);


	/*Block and Grid Dims*/
	dim3 blockDim(GEMM_TILE_SIZE, GEMM_TILE_SIZE);
	dim3 gridDim((cShape.width + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE, 1, cShape.count); // 128x1x3
	size_t sharedMemSize = 2 * GEMM_TILE_SIZE * GEMM_TILE_SIZE * sizeof(float);

	std::cout << "'Speed-up' COPY GPU Starting!\n";
	hipEvent_t start1, stop1;
	hipEventCreate(&start1); hipEventCreate(&stop1);
	hipEventRecord(start1);
	std::cout << "\tMemory Size: " << sharedMemSize << " Bytes! \n";
	auto start = std::chrono::high_resolution_clock::now();
	gemmLayer_gpu_speed << <gridDim, blockDim, sharedMemSize >> > (d_a, aShape, d_b, bShape, d_c, cShape);
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "GPU execution time: " << elapsed.count() << " seconds\n";
	//gemmLayer_gpu_v2 << <gridDim, blockDim >> > (d_a, aShape, d_b, bShape, d_c, cShape);
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	float ms; hipEventElapsedTime(&ms, start1, stop1);
	std::cout << "'Speed-up' COPY Batch " << BatchSize << " Time: " << ms << " ms\n";
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Kernel Launch Error: " << hipGetErrorString(err) << "\n";
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		std::cout << "CUDA Error: " << hipGetErrorString(err) << "\n";
	}

	hipMemcpy(h_c, d_c, tensorSize(cShape) * sizeof(float), hipMemcpyDeviceToHost);

	/* hipFree() functions */
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	/*CPU Free*/
	free(h_a);
	free(h_b);
	//free(h_c);

	return h_c;
}

int evaluateGpuGemm_copy (TensorShape aShape, TensorShape bShape,
	TensorShape& cShape, GemmLayerArgs args, uint32_t BatchSize) {

	//cShape = { BatchSize, 1, 1, 4096 };

	cShape.height = aShape.height;
	cShape.width = bShape.width;
	cShape.channels = aShape.channels;
	cShape.count = BatchSize;

	float* h_a = nullptr;
	float* h_b = nullptr;

	int retVal;
	retVal = makeTensor(&h_a, aShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		return -1;
	}
	retVal = makeTensor(&h_b, bShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		return -1;
	}

	/*C (out) Array initialization*/
	float* h_c = (float*)malloc(tensorSize(cShape) * sizeof(float));
	/*CUDA Malloc for in, out, filter and bias*/
	float* d_a, * d_b, * d_c;
	hipMalloc(&d_a, tensorSize(aShape) * sizeof(float));
	hipMalloc(&d_b, tensorSize(bShape) * sizeof(float));
	hipMalloc(&d_c, tensorSize(cShape) * sizeof(float));

	/*CUDA Memcpy for in, filter and bias*/
	hipMemcpy(d_a, h_a, tensorSize(aShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b , tensorSize(bShape) * sizeof(float), hipMemcpyHostToDevice);


	/*Block and Grid Dims*/
	dim3 blockDim(GEMM_TILE_SIZE,GEMM_TILE_SIZE);
	dim3 gridDim((cShape.width + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE, (cShape.height + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE);
	
	std::cout << "'Basic' COPY GPU Starting!\n";
	hipEvent_t start1, stop1;
	hipEventCreate(&start1); hipEventCreate(&stop1);
	hipEventRecord(start1);
	//std::cout << "Memory Size: " << sharedMemSize << " Bytes! \n";
	auto start = std::chrono::high_resolution_clock::now();
	gemmLayer_gpu << <gridDim, blockDim >> > (d_a, aShape, d_b, bShape, d_c, cShape);
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "GPU execution time: " << elapsed.count() << " seconds\n";
	
	//gemmLayer_gpu_v2 << <gridDim, blockDim >> > (d_a, aShape, d_b, bShape, d_c, cShape);
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	float ms; hipEventElapsedTime(&ms, start1, stop1);
	std::cout << "'Basic' COPY Batch " << BatchSize << " Time: " << ms << " ms\n";
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Kernel Launch Error: " << hipGetErrorString(err) << "\n";
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		std::cout << "CUDA Error: " << hipGetErrorString(err) << "\n";
	}
	
	hipMemcpy(h_c, d_c, tensorSize(cShape) * sizeof(float), hipMemcpyDeviceToHost);

	/* hipFree() functions */
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	/*CPU Free*/
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}

float* evaluateGpuGemm_copy2(TensorShape aShape, TensorShape bShape,
	TensorShape& cShape, GemmLayerArgs args, uint32_t BatchSize) {

	//cShape = { BatchSize, 1, 1, 4096 };

	cShape.height = aShape.height;
	cShape.width = bShape.width;
	cShape.channels = aShape.channels;
	cShape.count = BatchSize;

	float* h_a = nullptr;
	float* h_b = nullptr;

	int retVal;
	retVal = makeTensor(&h_a, aShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
	//	return -1;
	}
	retVal = makeTensor(&h_b, bShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
	//	return -1;
	}

	/*C (out) Array initialization*/
	float* h_c = (float*)malloc(tensorSize(cShape) * sizeof(float));
	/*CUDA Malloc for in, out, filter and bias*/
	float* d_a, * d_b, * d_c;
	hipMalloc(&d_a, tensorSize(aShape) * sizeof(float));
	hipMalloc(&d_b, tensorSize(bShape) * sizeof(float));
	hipMalloc(&d_c, tensorSize(cShape) * sizeof(float));

	/*CUDA Memcpy for in, filter and bias*/
	hipMemcpy(d_a, h_a, tensorSize(aShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, tensorSize(bShape) * sizeof(float), hipMemcpyHostToDevice);


	/*Block and Grid Dims*/
	dim3 blockDim(GEMM_TILE_SIZE, GEMM_TILE_SIZE);
	dim3 gridDim((cShape.width + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE, (cShape.height + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE);

	std::cout << "'Basic' COPY GPU Starting!\n";
	hipEvent_t start1, stop1;
	hipEventCreate(&start1); hipEventCreate(&stop1);
	hipEventRecord(start1);
	//std::cout << "Memory Size: " << sharedMemSize << " Bytes! \n";
	auto start = std::chrono::high_resolution_clock::now();
	gemmLayer_gpu << <gridDim, blockDim >> > (d_a, aShape, d_b, bShape, d_c, cShape);
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "GPU execution time: " << elapsed.count() << " seconds\n";

	//gemmLayer_gpu_v2 << <gridDim, blockDim >> > (d_a, aShape, d_b, bShape, d_c, cShape);
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	float ms; hipEventElapsedTime(&ms, start1, stop1);
	std::cout << "'Basic' COPY Batch " << BatchSize << " Time: " << ms << " ms\n";
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Kernel Launch Error: " << hipGetErrorString(err) << "\n";
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		std::cout << "CUDA Error: " << hipGetErrorString(err) << "\n";
	}

	hipMemcpy(h_c, d_c, tensorSize(cShape) * sizeof(float), hipMemcpyDeviceToHost);

	/* hipFree() functions */
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	/*CPU Free*/
	free(h_a);
	free(h_b);

	return h_c;
}

int evaluateGpuGemm_uvm(TensorShape aShape, TensorShape bShape,
	TensorShape& cShape, GemmLayerArgs args, uint32_t BatchSize) {

	//cShape = { BatchSize, 1, 1, 4096 };

	cShape.height = aShape.height;
	cShape.width = bShape.width;
	cShape.channels = aShape.channels;
	cShape.count = BatchSize;

	float* uvm_a = nullptr;
	float* uvm_b = nullptr;
	float* uvm_c = nullptr;

	/*CUDA UVM Babyyy*/
	hipMallocManaged(&uvm_a, aShape.count * aShape.width * sizeof(float));
	hipMallocManaged(&uvm_b, bShape.height * bShape.width * sizeof(float));
	hipMallocManaged(&uvm_c, cShape.count * cShape.width * sizeof(float));

	int retVal;
	retVal = makeTensor_uvm(&uvm_a, aShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		return -1;
	}
	retVal = makeTensor_uvm(&uvm_b, bShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		return -1;
	}

	/*Block and Grid Dims*/
	dim3 blockDim(GEMM_TILE_SIZE, GEMM_TILE_SIZE);
	dim3 gridDim((cShape.width + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE, 1, cShape.count);
	size_t sharedMemSize = 2 * GEMM_TILE_SIZE * GEMM_TILE_SIZE * sizeof(float);

	/*gpuGemm Kernel Call*/
	std::cout << "UVM GPU Starting!\n";
	hipEvent_t start1, stop1;
	hipEventCreate(&start1); hipEventCreate(&stop1);
	hipEventRecord(start1);
	auto start = std::chrono::high_resolution_clock::now();
	gemmLayer_gpu_v2 << <gridDim, blockDim, sharedMemSize >> > (uvm_a, aShape, uvm_b, bShape, uvm_c, cShape);
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "GPU execution time: " << elapsed.count() << " seconds\n";
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	float ms; hipEventElapsedTime(&ms, start1, stop1);
	std::cout << "UVM Batch " << BatchSize << " Time: " << ms << " ms\n";
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Kernel Launch Error: " << hipGetErrorString(err) << "\n";
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		std::cout << "CUDA Error: " << hipGetErrorString(err) << "\n";
	}
	
	hipFree(uvm_a);
	hipFree(uvm_b);
	hipFree(uvm_c);
	return 0;
}

float* evaluateGpuGemm_uvm2(TensorShape aShape, TensorShape bShape,
	TensorShape& cShape, GemmLayerArgs args, uint32_t BatchSize) {

	//cShape = { BatchSize, 1, 1, 4096 };

	cShape.height = aShape.height;
	cShape.width = bShape.width;
	cShape.channels = aShape.channels;
	cShape.count = BatchSize;

	float* uvm_a = nullptr;
	float* uvm_b = nullptr;
	float* uvm_c = nullptr;

	/*CUDA UVM Babyyy*/
	hipMallocManaged(&uvm_a, aShape.count * aShape.width * sizeof(float));
	hipMallocManaged(&uvm_b, bShape.height * bShape.width * sizeof(float));
	hipMallocManaged(&uvm_c, cShape.count * cShape.width * sizeof(float));

	int retVal;
	retVal = makeTensor_uvm(&uvm_a, aShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		//return -1;
	}
	retVal = makeTensor_uvm(&uvm_b, bShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n";
		//return 99;
	}

	/*Block and Grid Dims*/
	dim3 blockDim(GEMM_TILE_SIZE, GEMM_TILE_SIZE);
	dim3 gridDim((cShape.width + GEMM_TILE_SIZE - 1) / GEMM_TILE_SIZE, 1, cShape.count);
	size_t sharedMemSize = 2 * GEMM_TILE_SIZE * GEMM_TILE_SIZE * sizeof(float);

	/*gpuGemm Kernel Call*/
	std::cout << "UVM GPU Starting!\n";
	hipEvent_t start1, stop1;
	hipEventCreate(&start1); hipEventCreate(&stop1);
	hipEventRecord(start1);
	auto start = std::chrono::high_resolution_clock::now();
	gemmLayer_gpu_v2 << <gridDim, blockDim, sharedMemSize >> > (uvm_a, aShape, uvm_b, bShape, uvm_c, cShape);
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "GPU execution time: " << elapsed.count() << " seconds\n";
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	float ms; hipEventElapsedTime(&ms, start1, stop1);
	std::cout << "UVM Batch " << BatchSize << " Time: " << ms << " ms\n";
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "Kernel Launch Error: " << hipGetErrorString(err) << "\n";
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		std::cout << "CUDA Error: " << hipGetErrorString(err) << "\n";
	}

	hipFree(uvm_a);
	hipFree(uvm_b);
	return uvm_c;
}

//	STUDENT: Add functions here (No)
